#include "hip/hip_runtime.h"
#include "reactor.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <string>
#include <cmath>
#include <stdexcept>


/// Helper functions to navigate through a variadic number of fields at compile-time.

// Apply lambda to each element of a tuple (a compile-time ranged-based for loop).
template <std::size_t index = 0, typename Function, typename... T>
inline __host__ __device__ typename std::enable_if<index == sizeof...(T), void>::type
for_each(std::tuple<T...> &, Function) { }
template <std::size_t index = 0, typename Function, typename... T>
inline __host__ __device__ typename std::enable_if<index < sizeof...(T), void>::type
for_each(std::tuple<T...>& t, Function f)
{
    f(index, std::get<index>(t));
    for_each<index + 1, Function, T...>(t, f);
}

// Apply lambda function as a more conventional "for loop" (but at compile-time).
template <std::size_t... I, typename Function>
inline __host__ __device__ void for_each(std::index_sequence<I...>, Function func)
{
    (func(std::integral_constant<std::size_t, I>{}), ...);
}


/// Main execution on GPU device.

namespace kernel
{
    // Check CUDA for errors after GPU execution and throw them.
    __host__ void throw_errors()
    {
        hipError_t error = hipGetLastError();
        if (error != hipSuccess)
        {
            std::string message = "CUDA Kernel Error: "
                                + std::string(hipGetErrorString(error));
            throw CudaError(message);
        }
    }

    // Implementation is on a 2d grid with periodic boundary conditions.
    // GPU divided into an (tile_rows x tile_cols) tile (blocks) with
    // a CUDA thread for each tile sharing this memory. Varying the tile size
    // will potentially improve performance on different hardware - I found
    // 16x16 was close to optimum on my machine for simulations on a 1024x1024 grid.
    static constexpr int tile_rows = 16;
    static constexpr int tile_cols = 16;
    // We need ghost points for each tile so we can evaluate derivatives
    // (specifically the Laplacian for diffusion) at the tile borders.
    static constexpr int num_ghost = 1; // <- minimum for second-order finite-difference stencil.

    // Stencil parameters - 2d space (x, y), and time t.
    __constant__ Scalar dt, dxInv, dyInv; // size of each space-time point
    __constant__ int nrows, ncols;        // number of points in spatial grid

    // Diffusion coefficients for each species.
    static constexpr int MAX_FIELDS = 16;
    __constant__ Scalar D[MAX_FIELDS];


    /// Execution of chemical flux on CUDA device.

    // The chemical flux may take some number of constant parameters.
    // The implementation of the chemical flux is known at compile-time, so we can read
    // its function signature to determine the number via a bit of template metaprogramming.
    // This makes it trivial to implement new systems by just defining a new chemical
    // flux function. The implementation happens at compile-time, so should not have a
    // performance overhead.
    static constexpr int MAX_PARAMETERS = 256;
    __constant__ Scalar chemical_flux_parameters[MAX_PARAMETERS];

    // Evaluate chemical flux by unpacking the correct number of parameters to match
    // the implementation's signature.
    template <typename Implementation, std::size_t... I, typename... Fields>
    __device__ auto evaluate_chemical_flux(std::index_sequence<I...>, Fields&&... fields)
    {
        return Implementation::chemical_flux(std::forward<Fields>(fields)...,
                                             chemical_flux_parameters[I]...);
    }
    template <typename Implementation, typename... Fields>
    __device__ auto evaluate_chemical_flux(Fields&&... fields)
    {
        constexpr auto nparams = Implementation::nparams_device();
        return evaluate_chemical_flux<Implementation>(std::make_index_sequence<nparams>{},
                                                      std::forward<Fields>(fields)...);
    }

    // Extract the [i][j] element from each field in the tile.
    template<typename Tile, std::size_t... I>
    __device__ auto local_fields(Tile&& tile, std::index_sequence<I...>, int i, int j) {
        return std::make_tuple(tile[I][i][j]...);
    }


    /// The kernel itself.

    template <typename System, typename Fields>
    __global__ void reactor_integration(Fields fields)
    {
        constexpr size_t nfields = std::tuple_size<Fields>::value;
        static_assert(nfields == System::nfields(), "Number of fields passed incompatible with system!");

        // Global indices.
        const int row = blockIdx.y * blockDim.y + threadIdx.y;
        const int col = blockIdx.x * blockDim.x + threadIdx.x;
        const int index = col + row * ncols;

        // Local indices.
        const int i = threadIdx.y + num_ghost;
        const int j = threadIdx.x + num_ghost;

        // Load tile into shared memory.
        __shared__ Scalar tile[nfields][tile_rows + 2*num_ghost][tile_cols + 2*num_ghost];

        {
            auto load = [&](auto m, auto field)
            {
                tile[m][i][j] = field[index];
            };
            for_each(fields, load);
        }

        // Fill in ghost points.

        if (threadIdx.y < num_ghost)
        {
            auto load = [&](auto m, auto field)
            {
                tile[m][i - num_ghost][j] = field[col + ((row - num_ghost + nrows) % nrows) * ncols];
                tile[m][i + tile_rows][j] = field[col + ((row + tile_rows) % nrows) * ncols];
            };
            for_each(fields, load);
        }

        if (threadIdx.x < num_ghost)
        {
            auto load = [&](auto m, auto field)
            {
                tile[m][i][j - num_ghost] = field[(col - num_ghost + ncols) % ncols + row * ncols];
                tile[m][i][j + tile_cols] = field[(col + tile_cols) % ncols         + row * ncols];
            };
            for_each(fields, load);
        }

        if (threadIdx.x < num_ghost and threadIdx.y < num_ghost)
        {
            auto load = [&](auto m, auto field)
            {
                tile[m][i - num_ghost][j - num_ghost] = field[(col - num_ghost + ncols) % ncols + ((row - num_ghost + nrows) % nrows) * ncols];
                tile[m][i - num_ghost][j + tile_cols] = field[(col + tile_cols) % ncols         + ((row - num_ghost + nrows) % nrows) * ncols];
                tile[m][i + tile_rows][j - num_ghost] = field[(col - num_ghost + ncols) % ncols + ((row + tile_rows) % nrows) * ncols];
                tile[m][i + tile_rows][j + tile_cols] = field[(col + tile_cols) % ncols         + ((row + tile_rows) % nrows) * ncols];
            };
            for_each(fields, load);
        }

        __syncthreads();

        // Contributions to evolution equation from reactions.
        auto flux = [&](auto&&... args)
        {
            return evaluate_chemical_flux<System>(std::forward<decltype(args)>(args)...);
        };
        auto rhs = std::apply(flux, local_fields(tile, std::make_index_sequence<nfields>{}, i, j));

        // Contributions from Laplacian, making it into reaction-diffusion.
        // Implementation uses a basic second-order central finite difference stencil.
        auto diffusion = [&](auto m)
        {
            std::get<m>(rhs) += D[m] * (  dxInv*dxInv * (tile[m][i+1][j] + tile[m][i-1][j])
                                        + dyInv*dyInv * (tile[m][i][j+1] + tile[m][i][j-1])
                                        - 2*(dxInv*dxInv + dyInv*dyInv) * tile[m][i][j]);
        };
        for_each(std::make_index_sequence<nfields>{}, diffusion);

        // Integrate the original field with an Euler forward step and we're done.
        auto evolve = [&](auto m)
        {
            std::get<m>(fields)[index] += std::get<m>(rhs) * dt;
        };
        for_each(std::make_index_sequence<nfields>{}, evolve);
    }

    // Basic kernel to check for errors (e.g. if fields become nan or inf).
    template <typename System, typename Fields>
    __global__ void check_finite(Fields fields, bool* finite)
    {
        constexpr size_t nfields = std::tuple_size<Fields>::value;
        static_assert(nfields == System::nfields(), "Number of fields passed incompatible with system!");

        // Global indices.
        const int row = blockIdx.y * blockDim.y + threadIdx.y;
        const int col = blockIdx.x * blockDim.x + threadIdx.x;
        const int index = col + row * ncols;

        auto check = [&](auto m, auto field)
        {
            if (not std::isfinite(field[index])) *finite = false;
        };
        for_each(fields, check);
    }
}


/// Host device definitions.

template <typename System>
Reactor<System>::Reactor(Reactor<System>&& other) noexcept
    : dt(other.dt), dx(other.dx), dy(other.dy), dxInv(other.dxInv), dyInv(other.dyInv),
    nrows(other.nrows), ncols(other.ncols),
    pitch_width(other.pitch_width), mem_size(other.mem_size),
    D(std::move(other.D)), flux_parameters(other.flux_parameters),
    current_step(other.current_step), pitch(std::move(other.pitch)),
    fields(std::move(other.fields))
{
    // Zero the pointers in the rvalue so that no device memory is freed at deconstruction.
    auto zero = [&](auto m)
    {
        std::get<m>(other.fields) = nullptr;
    };
    for_each(std::make_index_sequence<nfields>{}, zero);
}

template <typename System>
Reactor<System>::Reactor(const InitialState& initial_fields,
                         Scalar dt, Scalar dx, Scalar dy,
                         std::array<Scalar, nfields> D,
                         std::array<Scalar, nparams_host> params,
                         int current_step)
    : dt(dt), dx(dx), dy(dy), dxInv(1/dx), dyInv(1/dy),
    nrows(std::get<0>(initial_fields).rows()),
    ncols(std::get<0>(initial_fields).cols()),
    pitch_width(std::get<0>(initial_fields).cols() * sizeof(Scalar)),
    mem_size(std::get<0>(initial_fields).rows() * std::get<0>(initial_fields).cols() * sizeof(Scalar)),
    D(D), flux_parameters(params), current_step(current_step)
{
    auto malloc = [&](auto m)
    {
        if (std::get<m>(initial_fields).rows() != nrows or std::get<1>(initial_fields).cols() != ncols)
            throw std::runtime_error("fields do not have the same dimensions!");

        // Initialize device memory.
        hipMallocPitch(&std::get<m>(fields), &pitch[m], pitch_width, nrows);
        set_field<m>(std::get<m>(initial_fields));
    };
    for_each(std::make_index_sequence<nfields>{}, malloc);

    kernel::throw_errors();
}

template <typename System>
Reactor<System>::~Reactor()
{
    auto free = [&](auto m)
    {
        hipFree(std::get<m>(fields));
    };
    for_each(std::make_index_sequence<nfields>{}, free);
}

template <typename System>
void Reactor<System>::run(const int nsteps)
{
    // Set parameters on device.
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::dt), &dt, sizeof(Scalar));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::dxInv), &dxInv, sizeof(Scalar));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::dyInv), &dyInv, sizeof(Scalar));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::nrows), &nrows, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::ncols), &ncols, sizeof(int));

    // Diffusion coefficients for each species.
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::D), &D, sizeof(D));

    // Extra system-dependent parameters for the chemical flux.
    auto params = device_parameters();
    hipMemcpyToSymbol(HIP_SYMBOL(kernel::chemical_flux_parameters), &params, sizeof(params));

    // Calculate new state on device.
    const dim3 block_dim(kernel::tile_cols, kernel::tile_rows);
    const dim3 grid_size((ncols + block_dim.x - 1) / block_dim.x,
                         (nrows + block_dim.y - 1) / block_dim.y);

    for (int step = 0; step < nsteps; ++step)
    {
        kernel::reactor_integration<System><<<grid_size, block_dim>>>(fields);
    }

    hipDeviceSynchronize();
    kernel::throw_errors();

    // Numerical errors in integration often cause fields to diverge or go to nan, so we
    // need to check for these on the device and raise them up the stack.
    bool finite{true}, *device_finite;
    hipMalloc(&device_finite, sizeof(bool));
    hipMemcpy(device_finite, &finite, sizeof(bool), hipMemcpyHostToDevice);
    kernel::check_finite<System><<<grid_size, block_dim>>>(fields, device_finite);
    hipMemcpy(&finite, device_finite, sizeof(bool), hipMemcpyDeviceToHost);

    if (not finite)
    {
        std::string message = "an unknown numerical error occurred during simulation";
        throw kernel::CudaError(message);
    }

    current_step += nsteps;
}

template <typename System>
State Reactor<System>::get_field(Scalar* field) const
{
    auto out = State(nrows, ncols);
    hipMemcpy(out.data(), field, mem_size, hipMemcpyDeviceToHost);
    return out;
}

template <typename System>
void Reactor<System>::set_field(const State& source, Scalar* destination)
{
    hipMemcpy(destination, source.data(), mem_size, hipMemcpyHostToDevice);
}

template <typename System>
int Reactor<System>::step() const
{
    return current_step;
}

template <typename System>
Scalar Reactor<System>::time() const
{
    return static_cast<Scalar>(current_step) * dt;
}


// Define the systems here so CUDA compiler (nvcc) knows to compile them.
template class Reactor<CellPolarisation>;
template class Reactor<ActiveModelB>;
template class Reactor<ToyModel>;